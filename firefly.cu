#include <iostream>
#include <vector>
#include <cmath>
#include <algorithm>
#include <random>
#include <limits>
#include <fstream>
#include <chrono>
#include <hip/hip_runtime.h>
// #include "/home/pp24/pp24s036/firefly/NVTX/c/include/nvtx3/nvtx3.hpp"


using namespace std;

// CUDA version of the fitness function (fun)

// double* arr[1024];
// 1 block, 1024 threads: 1024 elements
// warp_reduce: google
// 1 block, 32 ~ 128 threads: 1024 elements

// 1024 threads -> 32 warp -> CUDA -> 1 warp = 32 threads
// thread in a wrap can use register to swap data -> each wrap use wrap reduce -> each wrap has only 1 sum
// write 32 sum to shared memory
// let 1 wrap to read shared memory
// do wrap reduce, write back to the fitness
// swap data between wraps -> use shared memory

// N blocks, each block 1024 threads, block i deal with pop[i][0~D-1]


// [[...],
//  [...],
//  [...]]

// pop: N x D, fitness: N
__global__ static void fun_kernel(double* pop, double* fitness, int N, int D) {

    // Best practice: coalesced access
    // threads in wrap accesses continous memory
    // warp0 ( 0 ~ 31)
    // warp1 (32 ~ 63)

    // D = 1024
    // warp 0 ~ 32 -> 1024: 1024 x
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
    //     double funsum = 0.0;
    //     for (int j = 0; j < D; j++) {
    //         // thread 0: pop[0 ~ D]   0
    //         // thread 1: pop[D ~ 2D]  D
    //         double x = pop[idx * D + j];
    //         funsum += x * x - 10 * cos(2 * M_PI * x);
    //     }
    //     funsum += 10 * D;
    //     fitness[idx] = funsum;
        double funsum = 0.0;
        //printf("Thread info - idx: %d, blockIdx.x: %d, blockDim.x: %d, threadIdx.x: %d\n",idx, blockIdx.x, blockDim.x, threadIdx.x);
        printf("Thread info: idx = %d\n", idx);
        if (idx < 10) {
            printf("Thread info - idx: %d, blockIdx.x: %d, blockDim.x: %d, threadIdx.x: %d\n", idx, blockIdx.x, blockDim.x, threadIdx.x);
        }

            // Coalesced access: Each thread processes a column of `pop`
            for (int j = threadIdx.x; j < D; j += blockDim.x) {
                double x = pop[idx + j * N];  // Coalesced memory access
                funsum += x * x - 10 * cosf(2 * M_PI * x);
            }

            // Aggregate the result
            fitness[idx] = funsum + 10 * D;
    }
}

class FA {
public:
    FA(int dimen, int population, int max_iter)
        : D(dimen), N(population), it(max_iter), A(0.97), B(1.0), G(0.0001) {
        Ub.resize(D, 3.0);
        Lb.resize(D, 0.0);
    }

    vector<double> fun(const vector<double>& pop) {
        vector<double> fitness(N);

        // Allocate GPU memory
        double* d_pop;
        double* d_fitness;
        hipMalloc(&d_pop, N * D * sizeof(double));
        hipMalloc(&d_fitness, N * sizeof(double));

        // Copy data to GPU
        hipMemcpy(d_pop, pop.data(),N * D * sizeof(double), hipMemcpyHostToDevice);

        // Launch kernel
        int blockSize = 1024;
        int numBlocks = (N + blockSize - 1) / blockSize;

        // GPU A100 has 108SM, each SM can compute multi-block
        // 1 block -> SM
        std::cerr << "numBlocks: " << numBlocks << std::endl;
        fun_kernel<<<numBlocks, blockSize>>>(d_pop, d_fitness, N, D);
        hipDeviceSynchronize();
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        }
        // Copy results back to CPU
        hipMemcpy(fitness.data(), d_fitness, N * sizeof(double), hipMemcpyDeviceToHost);

        // Free GPU memory
        hipFree(d_pop);
        hipFree(d_fitness);
        hipDeviceReset(); // Clean up and flush device logs

        return fitness;
    }

    int D;                  // Dimension of problems
    int N;                  // Population size
    int it;                 // Max iteration
    vector<double> Ub;      // Upper bound
    vector<double> Lb;      // Lower bound
    double A;               // Strength
    double B;               // Attractiveness constant
    double G;               // Absorption coefficient
};

int main() {
    int dimen, population, max_iter;

    auto start_time = chrono::high_resolution_clock::now();

    random_device rd;
    mt19937 gen(0); // rd()
    uniform_real_distribution<> dis(-1024, 1024);

    FA fa(1024, 5, 5);
    vector<double> pop(fa.N * fa.D); // 1D array for population
    
    // Initialize population
    for (int i = 0; i < fa.N; i++) {
        for (int j = 0; j < fa.D; j++) {
            pop[i * fa.D + j] = dis(gen); // Linear indexing
        }
    }

    vector<double> fitness = fa.fun(pop);

    vector<double> best_list;
    vector<vector<double>> best_para_list;

    auto min_iter = min_element(fitness.begin(), fitness.end());
    best_list.push_back(*min_iter);
    int arr = distance(fitness.begin(), min_iter);

    // Extract the best parameters
    vector<double> best_para(fa.D);
    for (int j = 0; j < fa.D; j++) {
        best_para[j] = pop[arr * fa.D + j];
    }
    best_para_list.push_back(best_para);

    double best_iter;
    double best_ = numeric_limits<double>::max();
    vector<double> best_para_(fa.D);

    int it = 1;
    while (it < fa.it) {
        for (int i = 0; i < fa.N; i++) {
            for (int j = 0; j < fa.D; j++) {
                double steps = fa.A * (dis(gen) - 0.5) * abs(fa.Ub[0] - fa.Lb[0]);
                double r_distance = 0;

                for (int k = 0; k < fa.N; k++) {
                    if (fitness[i] > fitness[k]) {
                        r_distance += pow(pop[i * fa.D + j] - pop[k * fa.D + j], 2);
                        double Beta = fa.B * exp(-fa.G * r_distance);
                        double xnew = pop[i * fa.D + j] + Beta * (pop[k * fa.D + j] - pop[i * fa.D + j]) + steps;

                        xnew = min(max(xnew, fa.Lb[0]), fa.Ub[0]);
                        pop[i * fa.D + j] = xnew;

                        // Update fitness after position update
                        fitness = fa.fun(pop);
                        auto best_iter = min_element(fitness.begin(), fitness.end());
                        best_ = *best_iter;
                        int arr_ = distance(fitness.begin(), best_iter);

                        for (int j = 0; j < fa.D; j++) {
                            best_para_[j] = pop[arr_ * fa.D + j];
                        }
                    }
                }
            }
        }
        best_list.push_back(best_);
        best_para_list.push_back(best_para_);
        it++;
        cout << "Iteration " << it << " finished" << endl;
    }

    // Save results to file
    ofstream file("results_cuda.csv");
    if (file.is_open()) {
        // Write header
        file << "Dimension_1";
        for (int d = 1; d < fa.D; ++d) {
            file << ",Dimension_" << d + 1;
        }
        file << ",Fitness\n";

        // Write population matrix and fitness
        for (int i = 0; i < fa.N; ++i) {
            for (int j = 0; j < fa.D; ++j) {
                file << pop[i * fa.D + j];
                if (j < fa.D - 1) {
                    file << ",";
                }
            }
            file << "," << fitness[i] << "\n";
        }

        // Write best fitness values
        file << "\nGeneration,Best Fitness\n";
        for (int i = 0; i < best_list.size(); ++i) {
            file << i << "," << best_list[i] << "\n";
        }
        file.close();
        cout << "Results saved to results_cuda.csv" << endl;
    }

    auto end_time = chrono::high_resolution_clock::now();
    chrono::duration<double> elapsed_time = end_time - start_time;
    cout << "Program execution time: " << elapsed_time.count() << " seconds" << endl;

    return 0;
}
